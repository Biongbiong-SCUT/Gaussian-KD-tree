#include "hip/hip_runtime.h"
// we pass the tree by value, because we need to load most of it anyway

void gaussianSlice(float *positions, float *values, 
		   int nPositions, int samples, float sigma) {
    
    // grab the tree header
    kd_tree t;
    CUDA_SAFE_CALL(hipMemcpy(&t, tree, sizeof(kd_tree), hipMemcpyDeviceToHost));
    
    int roundedNPositions = ((nPositions-1)/LOOKUP_CHUNK_SIZE+1)*LOOKUP_CHUNK_SIZE;

    // copy the position array to the device
    float *devicePositions;
    CUDA_SAFE_CALL(hipMalloc((void **)&devicePositions,
			      sizeof(float)*roundedNPositions*t.positionDimensions));
    CUDA_SAFE_CALL(hipMemcpy(devicePositions, positions,
			      sizeof(float)*nPositions*t.positionDimensions,
			      hipMemcpyHostToDevice));

    // clear the debug array
    CUDA_SAFE_CALL(hipMemset(debugPtr, 0, sizeof(int)*1024*1024));
    
    // allocate the values array on the device
    float *deviceValues;
    CUDA_SAFE_CALL(hipMalloc((void **)&deviceValues, 
			      sizeof(float)*nPositions*t.valueDimensions));
    
    printf("Slicing... "); fflush(stdout);
    // do 4k samples at a time   
    int chunk = 1<<12;
    for (int i = 0; i < nPositions; i += chunk) {
	printf("."); fflush(stdout);
	if (i + chunk > nPositions) chunk = nPositions - i;
	int roundedChunk = ((chunk-1)/LOOKUP_CHUNK_SIZE + 1);
	uint4 seed = make_uint4(rand(), rand(), rand(), rand());
	_gaussianLookup<<<roundedChunk, LOOKUP_CHUNK_SIZE>>>(t, devicePositions + i*t.positionDimensions,
							     deviceValues + i*t.valueDimensions,
							     chunk, sigma, samples, seed, SLICE);
	CUT_CHECK_ERROR("gaussianLookup failed\n");
    }

    printf("Done\n"); fflush(stdout);

    // copy back the values
    CUDA_SAFE_CALL(hipMemcpy(values, deviceValues, 
			      sizeof(float)*nPositions*t.valueDimensions,
			      hipMemcpyDeviceToHost));
    
    // check the debug array
    /*
    int dbg[1024*20];
    CUDA_SAFE_CALL(hipMemcpy(dbg, debugPtr, sizeof(int)*1024, hipMemcpyDeviceToHost));
    for (int i = 0; i < 1024*20; i+=4) {
	printf("%7i \t%7i \t%7i \t%7i\n", dbg[i], dbg[i+1], dbg[i+2], dbg[i+3]);
    }
    fflush(stdout);
    */

    // clean up
    CUDA_SAFE_CALL(hipFree(devicePositions));
    CUDA_SAFE_CALL(hipFree(deviceValues));
}

